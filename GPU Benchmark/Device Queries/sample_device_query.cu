#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

main(){
int deviceCount, device;
//int gpuDeviceCount = 0;
struct hipDeviceProp_t properties;
hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
if (cudaResultCode != hipSuccess)
    deviceCount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < deviceCount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
    }
}
}
