#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
//#include "cuda_runtime_api.h"
//#define N 23040*10

__global__ void add( int *a, int *b, int *c ) {
 int tid = blockIdx.x*blockDim.x+threadIdx.x; // handle the data at this index
// if (tid < N)
 c[tid] = a[tid] + b[tid];
}

int main( void ) {
int deviceCount, device;
int B,T,N;
struct hipDeviceProp_t properties;
int *a, *b, *c;
struct timeval start,stop;
int *dev_a, *dev_b, *dev_c;
hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
if (cudaResultCode != hipSuccess)
    deviceCount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < deviceCount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
            B=properties.multiProcessorCount;
            T=properties.maxThreadsPerMultiProcessor;
            N=10*properties.multiProcessorCount * 10*properties.maxThreadsPerMultiProcessor;
    }
}
a=(int*)malloc(N * sizeof(int));
b=(int*)malloc(N * sizeof(int));
c=(int*)malloc(N * sizeof(int));

 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, N * sizeof(int) );
hipMalloc( (void**)&dev_b, N * sizeof(int) );
hipMalloc( (void**)&dev_c, N * sizeof(int) );
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<N; i++) {
 a[i] = -i;
 b[i] = i * i;
 }
hipMemcpy( dev_a, a, N * sizeof(int),hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice );
gettimeofday(&start,NULL);
 add<<<B*10,T*10>>>( dev_a, dev_b, dev_c );
gettimeofday(&stop,NULL);
 // copy the array 'c' back from the GPU to the CPU
hipMemcpy( c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
 // display the results
// for (int i=0; i<N; i++) {
// printf( "%d + %d = %d\n", a[i], b[i], c[i] );
// }
 // free the memory allocated on the GPU
printf("Time elapsed is:%ld\n", (stop.tv_usec-start.tv_usec));
 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return 0;
}

