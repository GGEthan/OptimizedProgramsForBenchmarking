#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define TOTAL_MEMORY 1
#define BLOCK_SIZE 104876
//__global__ void add( int *a, int *b, int *c ) {
// int tid = blockIdx.x*blockDim.x+threadIdx.x; // handle the data at this index
//   c[tid] = a[tid] + b[tid];
 
//}

int main( void ) {
double time_s;
long start_time,end_time;
char *a;
struct timeval start,stop;
char *dev_a;
a=(char*)malloc(BLOCK_SIZE * sizeof(char));
memset(a,'$',BLOCK_SIZE);
int k;
 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, BLOCK_SIZE * sizeof(char) );
gettimeofday(&start,NULL);
start_time=start.tv_sec*1000000 + start.tv_usec;//get start time
for(k=0;k<(TOTAL_MEMORY/BLOCK_SIZE);k++)
hipMemcpy( dev_a, a, BLOCK_SIZE * sizeof(char),hipMemcpyHostToDevice );
gettimeofday(&stop,NULL);
end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time

// add<<<blocks,threads>>>( dev_a, dev_b, dev_c );
time_s=end_time-start_time;
printf("Time taken: %lf",(time_s));
printf("Throughput: %lf GB/s",(double)(TOTAL_MEMORY)/(time_s*1000));
// free the memory allocated on the GPU
 hipFree( dev_a );
 return 0;
}

