#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define NUM_CYCLES 50

__global__ void add( int *a, int *b, int *c ) {
 int tid = blockIdx.x*blockDim.x+threadIdx.x; // handle the data at this index
   c[tid]=a[tid] + b[tid];
   a[tid]=a[tid] + b[tid];
   c[tid]=a[tid] + b[tid];

 
}

int main( void ) {
int deviceCount, device;
int blocks,threads,n;
double time_s;
long start_time,end_time;
struct hipDeviceProp_t properties;
int *a, *b, *c;
struct timeval start,stop;
int *dev_a, *dev_b, *dev_c;
hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
if (cudaResultCode != hipSuccess)
    deviceCount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < deviceCount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
            blocks=properties.multiProcessorCount;
            threads=properties.maxThreadsPerMultiProcessor;
            n=properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor;
    }
}
a=(int*)malloc(n * sizeof(int));
b=(int*)malloc(n * sizeof(int));
c=(int*)malloc(n * sizeof(int));

 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, n * sizeof(int) );
hipMalloc( (void**)&dev_b, n * sizeof(int) );
hipMalloc( (void**)&dev_c, n * sizeof(int) );
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<n; i++) {
 a[i] = 23434;
 b[i] = i * i;
 }
hipMemcpy( dev_a, a, n * sizeof(int),hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, n * sizeof(int),hipMemcpyHostToDevice );
gettimeofday(&start,NULL);
int l;
start_time=start.tv_sec*1000000 + start.tv_usec;//get start time
for(l=0;l<NUM_CYCLES;l++)
 add<<<blocks,threads>>>( dev_a, dev_b, dev_c );
gettimeofday(&stop,NULL);
end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time
 // copy the array 'c' back from the GPU to the CPU
hipMemcpy( c, dev_c, n * sizeof(int),hipMemcpyDeviceToHost );
 // display the results
// for (int i=0; i<N; i++) {
// printf( "%d + %d = %d\n", a[i], b[i], c[i] );
// }
 // free the memory allocated on the GPU
time_s=end_time-start_time;
printf("Time taken: %lf",time_s);
printf("GFLOPS: %lf",(double)(NUM_CYCLES*n*3)/(time_s*1000));

 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return 0;
}

